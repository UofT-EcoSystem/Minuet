#include <hipblas.h>

#include "minuet/cuda/helpers.cuh"

namespace minuet::cuda {

std::string CUBLASErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "Unknown Error";
}

#define MINUET_CHECK_CUBLAS(stmt, ...)                              \
  do {                                                              \
    auto return_code = (stmt);                                      \
    MINUET_CHECK(return_code == HIPBLAS_STATUS_SUCCESS,              \
                 "CUBLAS Error: ", CUBLASErrorString(return_code)); \
  } while (false)

template <typename T>
struct CUBLASDispatcher;

template <>
struct CUBLASDispatcher<half> {
  static constexpr const auto GEMM = hipblasHgemm;
  static constexpr const auto GEMMStridedBatched = hipblasHgemmStridedBatched;
};

template <>
struct CUBLASDispatcher<float> {
  static constexpr const auto GEMM = hipblasSgemm;
  static constexpr const auto GEMMStridedBatched = hipblasSgemmStridedBatched;
};

template <>
struct CUBLASDispatcher<double> {
  static constexpr const auto GEMM = hipblasDgemm;
  static constexpr const auto GEMMStridedBatched = hipblasDgemmStridedBatched;
};

template <typename FT>
void MatMul(std::size_t m,           //
            std::size_t k,           //
            std::size_t n,           //
            bool is_a_transposed,    //
            bool is_b_transposed,    //
            const FT *d_a,           //
            const FT *d_b,           //
            FT *d_c,                 //
            bool incremental,        //
            const Context &context,  //
            hipStream_t stream) {
  const FT alpha = 1.0f;
  const FT beta = incremental ? 1.0f : 0.0f;
  hipStream_t old_stream = nullptr;
  if (stream != nullptr) {
    MINUET_CHECK_CUBLAS(
        hipblasGetStream(context.GetCUBLASHandle(), &old_stream));
    MINUET_CHECK_CUBLAS(hipblasSetStream(context.GetCUBLASHandle(), stream));
  }
  // We need to swap a and b to get C^T = B^T A^T to receive row-major format
  MINUET_CHECK_CUBLAS(CUBLASDispatcher<FT>::GEMM(
      context.GetCUBLASHandle(),                    // handle
      is_b_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N,  // transa
      is_a_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N,  // transb
      n,                                            // m
      m,                                            // n
      k,                                            // k
      &alpha,                                       // alpha
      d_b,                                          // A
      is_b_transposed ? k : n,                      // lda
      d_a,                                          // B
      is_a_transposed ? m : k,                      // ldb
      &beta,                                        // beta
      d_c,                                          // C
      n                                             // ldc
      ));
  if (stream != nullptr) {
    MINUET_CHECK_CUBLAS(hipblasSetStream(context.GetCUBLASHandle(), old_stream));
  }
}

template <typename FT>
void BatchedMatMul(std::size_t b,           //
                   std::size_t m,           //
                   std::size_t k,           //
                   std::size_t n,           //
                   bool is_a_transposed,    //
                   bool is_b_transposed,    //
                   const FT *d_a,           //
                   const FT *d_b,           //
                   FT *d_c,                 //
                   bool incremental,        //
                   const Context &context,  //
                   hipStream_t stream) {
  if (b == 1) {
    MatMul(m,                //
           k,                //
           n,                //
           is_a_transposed,  //
           is_b_transposed,  //
           d_a,              //
           d_b,              //
           d_c,              //
           incremental,      //
           context,          //
           stream);
    return;
  }
  const FT alpha = 1.0f;
  const FT beta = incremental ? 1.0f : 0.0f;
  hipStream_t old_stream = nullptr;
  if (stream != nullptr) {
    MINUET_CHECK_CUBLAS(
        hipblasGetStream(context.GetCUBLASHandle(), &old_stream));
    MINUET_CHECK_CUBLAS(hipblasSetStream(context.GetCUBLASHandle(), stream));
  }
  MINUET_CHECK_CUBLAS(CUBLASDispatcher<FT>::GEMMStridedBatched(
      context.GetCUBLASHandle(),                    // handle
      is_b_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N,  // transa
      is_a_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N,  // transb
      n,                                            // m
      m,                                            // n
      k,                                            // k
      &alpha,                                       // alpha
      d_b,                                          // A
      is_b_transposed ? k : n,                      // lda
      k * n,                                        // strideA
      d_a,                                          // B
      is_a_transposed ? m : k,                      // ldb
      m * k,                                        // strideB
      &beta,                                        // beta
      d_c,                                          // C
      n,                                            // ldc
      m * n,                                        // strideC
      b                                             // batchCount
      ));
  if (stream != nullptr) {
    MINUET_CHECK_CUBLAS(hipblasSetStream(context.GetCUBLASHandle(), old_stream));
  }
}

#define MINUET_EXPLICIT_INSTANTIATE(FT)                                        \
  template void MatMul<FT>(                                                    \
      std::size_t m, std::size_t k, std::size_t n, bool is_a_transposed,       \
      bool is_b_transposed, const FT *d_a, const FT *d_b, FT *d_c,             \
      bool incremental, const Context &context, hipStream_t stream);          \
  template void BatchedMatMul<FT>(std::size_t b, std::size_t m, std::size_t k, \
                                  std::size_t n, bool is_a_transposed,         \
                                  bool is_b_transposed, const FT *d_a,         \
                                  const FT *d_b, FT *d_c, bool incremental,    \
                                  const Context &context, hipStream_t stream)
MINUET_EXPLICIT_INSTANTIATE(half);
MINUET_EXPLICIT_INSTANTIATE(float);
MINUET_EXPLICIT_INSTANTIATE(double);
#undef MINUET_EXPLICIT_INSTANTIATE

}  // namespace minuet::cuda