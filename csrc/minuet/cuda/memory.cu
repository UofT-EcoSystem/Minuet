#include "minuet/cuda/memory.cuh"

namespace minuet::cuda {

void *DirectMemoryFactory::Acquire(std::size_t size, std::size_t alignment) {
  void *d_data;
  MINUET_CHECK(alignment == 0 || CUDA_DEFAULT_ALIGNMENT % alignment == 0,
               "Cannot allocate CUDA memory with alignment ", alignment);
  MINUET_CHECK_CUDA(hipMalloc(&d_data, size));
  return d_data;
}

void DirectMemoryFactory::Release(void *data) {
  MINUET_CHECK_CUDA(hipFree(data));
}

void DirectMemoryFactory::SilentRelease(void *data) noexcept { hipFree(data); }

void *AsyncDirectMemoryFactory::Acquire(std::size_t size,
                                        std::size_t alignment) {
  MINUET_CHECK(alignment == 0 || CUDA_DEFAULT_ALIGNMENT % alignment == 0,
               "Cannot allocate CUDA memory with alignment ", alignment);
  void *d_data;
  MINUET_CHECK_CUDA(hipMallocAsync(&d_data, size, stream_));
  return d_data;
}

void AsyncDirectMemoryFactory::Release(void *data) {
  MINUET_CHECK_CUDA(hipFreeAsync(data, stream_));
}

void AsyncDirectMemoryFactory::SilentRelease(void *data) noexcept {
  hipFreeAsync(data, stream_);
}

}  // namespace minuet::cuda