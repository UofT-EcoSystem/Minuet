#include "minuet/cuda/event.cuh"

namespace minuet::cuda {

void Event::Record(hipStream_t stream) {
  MINUET_CHECK(!closed_, "Event is invalid");
  MINUET_CHECK_CUDA(hipEventRecord(event_, stream));
}

void Event::Synchronize() {
  MINUET_CHECK(!closed_, "Event is invalid");
  MINUET_CHECK_CUDA(hipEventSynchronize(event_));
}

float Event::Elapsed(const Event &after) const {
  float result;
  MINUET_CHECK(!closed_, "Event is invalid");
  MINUET_CHECK_CUDA(hipEventElapsedTime(&result, event_, after.event()));
  return result;
}

void Event::Close() {
  if (!closed_) {
    MINUET_CHECK_CUDA(hipEventDestroy(event_));
    closed_ = true;
  }
}

void Event::CloseNoExcept() noexcept {
  if (!closed_) {
    hipEventDestroy(event_);
    closed_ = true;
  }
}

}  // namespace minuet::cuda